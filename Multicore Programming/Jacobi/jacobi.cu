#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <cstdio>
#include "error_checks.h"

// Change this to 0 if CPU reference result is not needed
#define COMPUTE_CPU_REFERENCE 1
#define MAX_ITERATIONS 3000
#define GPU_VERSION 1 // TODO: Change this to 1 to enable the GPU version

// CPU kernel
void sweepCPU(double* phi, const double *phiPrev, const double *source, 
              double h2, int N)
{ 
    int i, j;
    int index, i1, i2, i3, i4;

    for (j = 1; j < N-1; j++) {
        for (i = 1; i < N-1; i++) {
            index = i + j*N; 
            i1 = (i-1) +   j   * N;
            i2 = (i+1) +   j   * N;
            i3 =   i   + (j-1) * N;
            i4 =   i   + (j+1) * N;
            phi[index] = 0.25 * (phiPrev[i1] + phiPrev[i2] + 
                                 phiPrev[i3] + phiPrev[i4] - 
                                 h2 * source[index]);
        } 
    }
} 

// GPU kernel
__global__
void sweepGPU(double *phi, const double *phiPrev, const double *source, 
              double h2, int N)
{
    // TODO: Add here the GPU implementation
}


double compareArrays(const double *a, const double *b, int N)
{
    double error = 0.0;
    int i;
    for (i = 0; i < N*N; i++) {
        error += fabs(a[i] - b[i]);
    }
    return error/(N*N);
}


double diffCPU(const double *phi, const double *phiPrev, int N)
{
    int i;
    double sum = 0;
    double diffsum = 0;
    
    for (i = 0; i < N*N; i++) {
        diffsum += (phi[i] - phiPrev[i]) * (phi[i] - phiPrev[i]);
        sum += phi[i] * phi[i];
    }

    return sqrt(diffsum/sum);
}

double diffGPU(const double *phi, const double *phiPrev, int N)
{
    // TODO: Add here the GPU implementation

}


int main() 
{ 
    timeval t1, t2; // Structs for timing
    const int N = 512;
    double h = 1.0 / (N - 1);
    int iterations;
    const double tolerance = 5e-4; // Stopping condition
    int i, j, index;

    const int blocksize = 16;
  
    double *phi      = new double[N*N]; 
    double *phiPrev  = new double[N*N]; 
    double *source   = new double[N*N]; 
    double *phi_cuda = new double[N*N]; 

    double *phi_d, *phiPrev_d, *source_d; 
    // Size of the arrays in bytes
    const int size = N*N*sizeof(double); 
    double diff;
  
    // Source initialization
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {      
            double x, y;
            x = (i - N / 2) * h;
            y = (j - N / 2) * h;
            index = j + i * N;
            if (((x - 0.25) * (x - 0.25) + y * y) < 0.1 * 0.1)
                source[index] = 1e10*h*h;
            else if (((x + 0.25) * (x + 0.25) + y * y) < 0.1 * 0.1)
                source[index] = -1e10*h*h;
            else
                source[index] = 0.0;
        }            
    }

    CUDA_CHECK( hipMalloc( (void**)&source_d, size) ); 
    CUDA_CHECK( hipMemcpy(source_d, source, size, hipMemcpyHostToDevice) ); 

    // Reset values to zero
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {      
            index = j + i * N;
            phi[index] = 0.0; 
            phiPrev[index] = 0.0; 
        }            
    }

    CUDA_CHECK( hipMalloc( (void**)&phi_d, size) ); 
    CUDA_CHECK( hipMalloc( (void**)&phiPrev_d, size) ); 
    CUDA_CHECK( hipMemcpy(phi_d, phi, size, hipMemcpyHostToDevice) );
    CUDA_CHECK( hipMemcpy(phiPrev_d, phiPrev, size, hipMemcpyHostToDevice) );

    // CPU version 
    if(COMPUTE_CPU_REFERENCE) { 
        gettimeofday(&t1, NULL);

        // Do sweeps untill difference is under the tolerance
        diff = tolerance * 2;
        iterations = 0;
        while (diff > tolerance && iterations < MAX_ITERATIONS) {
            sweepCPU(phiPrev, phi, source, h * h, N);
            sweepCPU(phi, phiPrev, source, h * h, N);
            
            iterations += 2;
            if (iterations % 100 == 0) {
                diff = diffCPU(phi, phiPrev, N);
                printf("%d %g\n", iterations, diff);
            }
        }
        gettimeofday(&t2, NULL);
        printf("CPU Jacobi: %g seconds, %d iterations\n", 
               t2.tv_sec - t1.tv_sec + 
               (t2.tv_usec - t1.tv_usec) / 1.0e6, iterations);
    }


#if GPU_VERSION
    // GPU version

    dim3 dimBlock(blocksize, blocksize); 
    dim3 dimGrid((N + blocksize - 1) / blocksize, (N + blocksize - 1) / blocksize); 
    
    //do sweeps until diff under tolerance
    diff = tolerance * 2;
    iterations = 0;

    gettimeofday(&t1, NULL);

    while (diff > tolerance && iterations < MAX_ITERATIONS) {
        // See above how the CPU update kernel is called
        // and implement similar calling sequence for the GPU code

        //// Add routines here
        // TODO: Add GPU kernel calls here (see CPU version above)

        iterations += 2;
        
        if (iterations % 100 == 0) {
            // TODO: Add GPU kernel calls here (see CPU version above)
            CHECK_ERROR_MSG("Difference computation");
            printf("%d %g\n", iterations, diff);
        }
    }
    
    //// Add here the routine to copy back the results
    //TODO: Copy back the results

    gettimeofday(&t2, NULL);
    printf("GPU Jacobi: %g seconds, %d iterations\n", 
           t2.tv_sec - t1.tv_sec + 
           (t2.tv_usec - t1.tv_usec) / 1.0e6, iterations);

    //// Add here the clean up code for all allocated CUDA resources
    // TODO: Add here the clean up code
#endif

    if (COMPUTE_CPU_REFERENCE) {
        printf("Average difference is %g\n", compareArrays(phi, phi_cuda, N));
    }
    
    delete[] phi; 
    delete[] phi_cuda;
    delete[] phiPrev; 
    delete[] source; 
    
    return EXIT_SUCCESS; 
} 