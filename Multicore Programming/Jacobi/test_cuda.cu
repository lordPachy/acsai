#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);

    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("Number of CUDA devices: %d\n", count);
    return 0;
}

